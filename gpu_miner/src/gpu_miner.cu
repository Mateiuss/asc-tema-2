#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "../include/utils.cuh"
#include <string.h>
#include <stdlib.h>
#include <inttypes.h>

// TODO: Implement function to search for all nonces from 1 through MAX_NONCE (inclusive) using CUDA Threads
__global__ void findNonce(BYTE *difficulty, BYTE *block_content, size_t current_length, uint32_t *nonce) {
	uint32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (thread_id > MAX_NONCE || *nonce != 0) {
		return;
	}

	char nonce_string[NONCE_SIZE];
	BYTE my_block_content[BLOCK_SIZE];
	intToString(thread_id, nonce_string);
	d_strcpy((char*)my_block_content, (char*)block_content);
	d_strcpy((char*)my_block_content + current_length, nonce_string);

	BYTE block_hash[SHA256_HASH_SIZE];

	apply_sha256(my_block_content, d_strlen((const char*)my_block_content), block_hash, 1);

	if (compare_hashes(block_hash, difficulty) <= 0) {
		*nonce = thread_id;
	}
}

int main(int argc, char **argv) {
	BYTE hashed_tx1[SHA256_HASH_SIZE], hashed_tx2[SHA256_HASH_SIZE], hashed_tx3[SHA256_HASH_SIZE], hashed_tx4[SHA256_HASH_SIZE],
			tx12[SHA256_HASH_SIZE * 2], tx34[SHA256_HASH_SIZE * 2], hashed_tx12[SHA256_HASH_SIZE], hashed_tx34[SHA256_HASH_SIZE],
			tx1234[SHA256_HASH_SIZE * 2], top_hash[SHA256_HASH_SIZE], block_content[BLOCK_SIZE];
	BYTE block_hash[SHA256_HASH_SIZE] = "0000000000000000000000000000000000000000000000000000000000000000"; // TODO: Update
	uint32_t nonce = 0; // TODO: Update
	size_t current_length;

	// Top hash
	apply_sha256(tx1, strlen((const char*)tx1), hashed_tx1, 1);
	apply_sha256(tx2, strlen((const char*)tx2), hashed_tx2, 1);
	apply_sha256(tx3, strlen((const char*)tx3), hashed_tx3, 1);
	apply_sha256(tx4, strlen((const char*)tx4), hashed_tx4, 1);
	strcpy((char *)tx12, (const char *)hashed_tx1);
	strcat((char *)tx12, (const char *)hashed_tx2);
	apply_sha256(tx12, strlen((const char*)tx12), hashed_tx12, 1);
	strcpy((char *)tx34, (const char *)hashed_tx3);
	strcat((char *)tx34, (const char *)hashed_tx4);
	apply_sha256(tx34, strlen((const char*)tx34), hashed_tx34, 1);
	strcpy((char *)tx1234, (const char *)hashed_tx12);
	strcat((char *)tx1234, (const char *)hashed_tx34);
	apply_sha256(tx1234, strlen((const char*)tx34), top_hash, 1);

	// prev_block_hash + top_hash
	strcpy((char*)block_content, (const char*)prev_block_hash);
	strcat((char*)block_content, (const char*)top_hash);
	current_length = strlen((char*) block_content);

	hipEvent_t start, stop;
	startTiming(&start, &stop);

	int threadsPerBlock = 512;
	int blocks = (int)(MAX_NONCE) / threadsPerBlock;

	if ((int)(MAX_NONCE) % threadsPerBlock != 0) {
		blocks++;
	}

	BYTE *device_block_content;
	hipMalloc(&device_block_content, BLOCK_SIZE);
	hipMemcpy(device_block_content, block_content, BLOCK_SIZE, hipMemcpyHostToDevice);

	BYTE *difficulty;
	hipMalloc(&difficulty, SHA256_HASH_SIZE);
	hipMemcpy(difficulty, DIFFICULTY, SHA256_HASH_SIZE, hipMemcpyHostToDevice);

	uint32_t *device_nonce;
	hipMalloc(&device_nonce, sizeof(uint32_t));
	hipMemset(device_nonce, 0, sizeof(uint32_t));

	findNonce<<<blocks, threadsPerBlock>>>(difficulty, device_block_content, current_length, device_nonce);

	hipDeviceSynchronize();

	char nonce_string[NONCE_SIZE];
	hipMemcpy(&nonce, device_nonce, sizeof(uint32_t), hipMemcpyDeviceToHost);
	sprintf(nonce_string, "%u", nonce);
	strcpy((char*)block_content + current_length, nonce_string);
	apply_sha256(block_content, strlen((const char*)block_content), block_hash, 1);
	
	float seconds = stopTiming(&start, &stop);
	printResult(block_hash, nonce, seconds);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
		return 1;
	}

	hipFree(device_block_content);
	hipFree(difficulty);
	hipFree(device_nonce);

	return 0;
}
